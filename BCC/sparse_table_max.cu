#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "include/sparse_table_max.cuh"
#define LOCAL_BLOCK_SIZE 100

#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA Error in %s at %s:%d: %s\n", #call, __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

using namespace std;

__inline__ __device__ int log2_int(int x) {
    return __float2int_rz(log2f(x));
}

__global__ void computeblocks_max(int* d_na, int m , int* d_a, int n){
    //store maximum values of local blocks
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i>=0 && i < m){
        int maxval = INT_MIN;
        for(int j=0; j<LOCAL_BLOCK_SIZE; j++){
            int index = i*LOCAL_BLOCK_SIZE + j;
            if(index < n){
                maxval = max(maxval , d_a[index]);
            }
        }
        d_na[i] = maxval;
    }
}



__global__ void preprocess_init_max(int* d_a, int* d_lookupmax, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        d_lookupmax[i] = d_a[i];
    }
}

__global__ void build_sparse_table_max(int* d_lookupmax, int k, int i, int threads) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < threads) {
        int index = j + i * k;
        int half_interval = 1 << (i - 1);
        d_lookupmax[index] = max(d_lookupmax[j + (i - 1) * k], d_lookupmax[j + (i - 1) * k + half_interval]);
    }
}

__global__ void query_sol_max(int* d_lookupmax, int* d_qleft, int* d_qright, int m, int q, int* d_ansmax, int* d_a , int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < q) {
        int a = d_qleft[i];
        int b = d_qright[i];
        int b_l = b / LOCAL_BLOCK_SIZE;
        int a_l = a / LOCAL_BLOCK_SIZE;
        int n_l,n_r;
        if(a_l == b_l){
            int maxval = INT_MIN;
            for(int j=a; j<=b; j++){
                maxval = max(maxval , d_a[j]);
            }
            d_ansmax[i] = maxval;
        }
        else if(a_l + 1 == b_l){
            int maxval = INT_MIN;
            for(int j=a; j<=b; j++){
                maxval = max(maxval , d_a[j]);
            }
            d_ansmax[i] = maxval;   
        }
        else{
            n_l = a_l + 1;
            n_r = b_l - 1;
            int maxval = INT_MIN;
            for(int j=a; j<(a_l+1)*LOCAL_BLOCK_SIZE; j++){
                maxval = max(maxval , d_a[j]);
            }
            for(int j=b_l*LOCAL_BLOCK_SIZE; j<=b; j++){
                maxval = max(maxval , d_a[j]);
            }
            int len = n_r - n_l + 1;
            int l = (int)log2_int(len);
            int index1 = n_l + l * m;
            int index2 = n_r - (1 << l) + 1 + l * m;
            maxval = max(maxval , max(d_lookupmax[index1], d_lookupmax[index2]));
            d_ansmax[i] = maxval;
        }
    }
}


void solveQ_max(int m, int* d_na, int q, int* d_left, int* d_right, int* d_ansmax , int* d_a , int n) {
    // mytimer mt{};

    int* d_lookupmax;
    int k = (int)log2(m);

    size_t lookupmax_size = (k + 1) * m * sizeof(int);
    CUDA_CHECK(hipMalloc(&d_lookupmax, lookupmax_size));

    //mt.timetaken_reset("alloc", 0);

    int blocks = (m + 1023) / 1024;
    preprocess_init_max<<<blocks, 1024>>>(d_na, d_lookupmax, m);
    CUDA_CHECK(hipDeviceSynchronize());

    int len = 1;
    for (int i = 1; i <= k; i++) {
        len *= 2;
        int threads = m - len + 1;
        blocks = (threads + 1023) / 1024;
        build_sparse_table_max<<<blocks, 1024>>>(d_lookupmax, m, i, threads);
        CUDA_CHECK(hipDeviceSynchronize());
    }

    //mt.timetaken_reset("built table", 1);

    blocks = (q + 1023) / 1024;
    query_sol_max<<<blocks, 1024>>>(d_lookupmax, d_left, d_right, m, q, d_ansmax, d_a , n);
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipFree(d_lookupmax));
}

void main_max( int n, int q, int* d_a , int* d_left , int* d_right , int* d_ansmax , int n_asize , int* d_na) {
    
    int blocks = (n_asize + 1023) / 1024;
    computeblocks_max<<<blocks, 1024>>>(d_na, n_asize, d_a, n);
    CUDA_CHECK(hipDeviceSynchronize());

    solveQ_max(n_asize, d_na, q, d_left, d_right, d_ansmax , d_a , n);

}
