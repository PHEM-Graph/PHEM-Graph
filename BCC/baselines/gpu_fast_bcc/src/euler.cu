#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

#include "bcc_memory_utils.cuh"
#include "list_ranking.cuh"
#include "cuda_utility.cuh"

// #define DEBUG

__global__ 
void create_dup_edges(
    int *d_edges_to, 
    int *d_edges_from, 
    const uint64_t *d_edges_input, 
    const int root,
    int N) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (thid < N) {

        if (thid == root)
          return;

        int edge_count = N - 1;
        uint64_t i = d_edges_input[thid];

        int u = i >> 32;  // Extract higher 32 bits
        int v = i & 0xFFFFFFFF; // Extract lower 32 bits
        
        int afterRoot = thid > root;
        // printf("For thid: %d, thid - afterRoot: %d, thid - afterRoot + edge_count: %d\n", thid, thid - afterRoot, thid - afterRoot + edge_count);

        d_edges_from[thid - afterRoot + edge_count] = d_edges_to[thid - afterRoot] = v;
        d_edges_to[thid - afterRoot + edge_count] = d_edges_from[thid - afterRoot] = u;
    }
}


__global__
void update_first_last_nxt(int* d_edges_from, int* d_edges_to, int* d_first, int* d_last, int* d_next, uint64_t* d_index, int E) {
    
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    if(thid < E) {
        int f = d_edges_from[d_index[thid]];
        int t = d_edges_to[d_index[thid]];

        if (thid == 0) {
            d_first[f] = d_index[thid];
            return;
        }

        if(thid == E - 1) {
            d_last[f] = d_index[thid];
        }

        int pf = d_edges_from[d_index[thid - 1]];
        int pt = d_edges_to[d_index[thid - 1]];

        // printf("For tid: %d, f: %d, t: %d, pf: %d, pt: %d\n", thid, f, t, pf, pt);

        // calculate the offset array
        if (f != pf) {
            d_first[f] = d_index[thid];
            // printf("d_last[%d] = d_index[%d] = %d\n", pf, thid - 1, d_index[thid - 1]);
            d_last[pf] = d_index[thid - 1];
        } else {
            d_next[d_index[thid - 1]] = d_index[thid];
        }
    }
}

__global__ 
void cal_succ(int* succ, const int* d_next, const int* d_first, const int* d_edges_from, int E) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    if (thid < E) {
        int revEdge = (thid + E / 2) % E;

        if (d_next[revEdge] == -1) {
            succ[thid] = d_first[d_edges_from[revEdge]];
        } else {
            succ[thid] = d_next[revEdge];
        }
    }
}

__global__ 
void break_cycle_kernel(int *d_last, int *d_succ, int* d_roots, int roots_count, int E) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < roots_count) {
        int root = d_roots[idx];
        // printf("Root: %d\n", root);
        if (d_last[root] != -1) {
            int last_edge = d_last[root];
            int rev_edge = (last_edge + E / 2) % E;
            // printf("\nFor root: %d, last_edge: %d, rev_edge: %d\n", root, last_edge, rev_edge);
            // Set the successor of the last edge to point to itself
            d_succ[rev_edge] = -1;
        }
    }
}

__global__
void find_parent(int E, int *rank, int *d_edges_to, int *d_edges_from, int *parent) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < E) {
        int f = d_edges_from[tid];
        int t = d_edges_to[tid];
        int rev_edge = (tid + E / 2) % E;
        // printf("for tid: %d, f: %d, t: %d, rev_edge: %d\n", tid, f, t, rev_edge);
        if(rank[tid] > rank[rev_edge]) {
            parent[t] = f;
        }
        else {
            parent[f] = t;
        }
    }
}

__global__ 
void merge_key_value(const int *arrayU, const int *arrayV, uint64_t *arrayE, uint64_t *d_indices, long size) {
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        // Cast to int64_t to ensure the shift operates on 64 bits
        uint64_t u = arrayU[idx];
        uint64_t v = arrayV[idx];

        arrayE[idx] = (u << 32) | (v & 0xFFFFFFFFLL);

        d_indices[idx] = idx;
    }
}

__global__
void compute_level_kernel(
    int* devRank, int* d_edges_from, int* d_edges_to, 
    int* d_parent, int* devW1Sum, int N, int E) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // if(idx == 0) {
    //     for(int i = 0; i < E; ++i) {
    //         printf("devW1Sum: %d\n", devW1Sum[i]);
    //     }
    // }

    if (idx < E) {
        int loc = E - 1 - devRank[idx];
        // idx is the edge number, so lets retrive the edge first
        int u = d_edges_from[idx];
        int v = d_edges_to[idx];
        
        if(d_parent[v] == u)
            devW1Sum[loc] = 1;
        else
            devW1Sum[loc] = -1;
    }
}

__global__
void finalise_level_kernel(
    int* d_edges_from, int* d_edges_to, 
    int* d_parent, 
    int* devRank, 
    int* d_prefix_sum, 
    int* d_level, 
    int E) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < E) {
        int loc = E - 1 - devRank[idx];
        int u = d_edges_from[idx];
        int v = d_edges_to[idx];

        // (p(V)，v)
        if(d_parent[v] == u) {
            //printf("u: %d, v: %d, d_prefix_sum[%d]: %d \n", u, v, idx, d_prefix_sum[idx]);
            d_level[v] = d_prefix_sum[loc];
        }
    }
}

void LexSortIndices(int* d_keys, int* d_values, uint64_t* d_indices_sorted, int num_items) {

    uint64_t *d_merged, *d_merged_keys_sorted;
    hipMalloc(&d_merged, sizeof(uint64_t) * num_items);
    hipMalloc(&d_merged_keys_sorted, sizeof(uint64_t) * num_items);

    uint64_t* d_indices;
    hipMalloc(&d_indices, sizeof(uint64_t)* num_items);   

    int blockSize = 1024;
    int numBlocks = (num_items + blockSize - 1) / blockSize; 

    // Initialize indices to 0, 1, 2, ..., num_items-1 also here
    merge_key_value<<<numBlocks, blockSize>>>(
        d_keys, 
        d_values, 
        d_merged, 
        d_indices, 
        num_items);
    CUCHECK(hipDeviceSynchronize());

    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    
    // Determine temporary storage requirements
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_merged, d_merged_keys_sorted, d_indices, d_indices_sorted, num_items);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Sort indices based on keys
    hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_merged, d_merged_keys_sorted, d_indices, d_indices_sorted, num_items);

    hipFree(d_merged);
    hipFree(d_merged_keys_sorted);
    hipFree(d_indices);
    hipFree(d_temp_storage);
}

void compute_level(
    int* devRank, int* d_edges_from, int* d_edges_to, 
    int* d_parent, int* devW1Sum, int* d_level, int N, int E) {

    int blockSize = 1024;
    int numBlocks = (E + blockSize - 1) / blockSize;

    compute_level_kernel<<<numBlocks, blockSize>>>(
        devRank, 
        d_edges_from, 
        d_edges_to, 
        d_parent, 
        devW1Sum, 
        N, E);

    CUCHECK(hipDeviceSynchronize());

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    int* d_level_info;
    CUCHECK(hipMalloc(&d_level_info, sizeof(int)*E));

    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, devW1Sum, d_level_info, E);
    // Allocate temporary storage
    CUCHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));
    // Run inclusive prefix sum
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, devW1Sum, d_level_info, E);
    CUCHECK(hipDeviceSynchronize());

    // devRank contains the prefix sum result
    finalise_level_kernel<<<numBlocks, blockSize>>>(
        d_edges_from, d_edges_to, 
        d_parent, devRank, d_level_info,
        d_level, E);
    CUCHECK(hipDeviceSynchronize());
}

__global__
void compute_first_last_occ(
    int* d_edges_from, 
    int* d_edges_to, 
    int* d_first_occ, 
    int* d_last_occ, 
    int* d_level,
    int* devRank, 
    int E) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < E) {
        int u = d_edges_to[idx];
        int v = d_edges_from[idx];       
        if (d_level[u] > d_level[v]) {
            d_last_occ[u] = devRank[idx]+1;
        } else {
            d_first_occ[v] = devRank[idx]+1;
        }
    }
}

__global__ 
void init_arrays(
    int *d_next, int *d_parent, int *d_level,
    int *d_first, int *d_last, 
    int* d_first_occ, int* d_last_occ, 
    int root, int E, int N) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < E) {
        // Initialize d_next array
        d_next[idx] = -1;  // You can modify this initialization as per your requirement
    }

    if (idx < N) {
        // Initialize d_parent, d_first, and d_last arrays
        d_parent[idx] = idx;
        d_first[idx] = -1;
        d_last[idx] = -1;
        d_level[idx] = 0;

        if (idx == root) {
            d_first_occ[idx] = 0;
            d_last_occ[idx] = 2 * N - 1;
        } else {
            d_first_occ[idx] = 2 * E;
            d_last_occ[idx] = -1;
        }
    }
}

void cuda_euler_tour(
    int N, 
    int root,
    GPU_BCG& g_bcg_ds) {
    
    uint64_t* d_edges_input = g_bcg_ds.d_parentEdge; 
    int E = N * 2 - 2;
    int roots_count = 1;

    int *d_edges_to = g_bcg_ds.d_edges_to;
    int *d_edges_from = g_bcg_ds.d_edges_from;
    
    // index can be considered as edge_num
    uint64_t *d_index =     g_bcg_ds.d_index;
    int *d_next       =     g_bcg_ds.d_next;
    int *d_roots      =     g_bcg_ds.d_roots;
    int *d_parent     =     g_bcg_ds.d_parent;
    int *d_level      =     g_bcg_ds.d_level;
    int *d_first      =     g_bcg_ds.d_first;
    int *d_last       =     g_bcg_ds.d_last;
    int *d_first_occ  =     g_bcg_ds.d_first_occ;
    int *d_last_occ   =     g_bcg_ds.d_last_occ;

    CUDA_CHECK(hipMemcpy(d_roots, &root, sizeof(int), hipMemcpyHostToDevice), "Failed to copy root");

    int blockSize = 1024;
    int numBlocks = (E + blockSize - 1) / blockSize;

    auto start = std::chrono::high_resolution_clock::now();

    init_arrays<<<numBlocks, blockSize>>>(
        d_next, 
        d_parent,
        d_level, 
        d_first, 
        d_last,
        d_first_occ,
        d_last_occ,
        root,
        E, N);

    CUDA_CHECK(hipDeviceSynchronize(), "Failed to synchronize init_arrays kernel");

    numBlocks = (N - 1 + blockSize - 1) / blockSize; 
    // Launch the kernel
    create_dup_edges<<<numBlocks, blockSize>>>(
        d_edges_to, 
        d_edges_from, 
        d_edges_input, 
        root, 
        N);

    CUDA_CHECK(hipDeviceSynchronize(), "Failed to synchronize create_dup_edges kernel");
    
    #ifdef DEBUG
        std::cout << "Printing from Euler Tour after creating duplicates:\n";
        DisplayDeviceEdgeList(d_edges_from, d_edges_to, E);
    #endif

    numBlocks = (E + blockSize - 1) / blockSize;

    LexSortIndices(d_edges_from, d_edges_to, d_index, E);

    #ifdef DEBUG
        std::cout << "Index array:\n";
        print_device_array(d_index, E);

        std::vector<int> sorted_from(E), sorted_to(E);
        std::vector<uint64_t> sorted_index(E);
        
        CUCHECK(hipMemcpy(sorted_index.data(), d_index, sizeof(uint64_t) * E, hipMemcpyDeviceToHost));
        CUCHECK(hipMemcpy(sorted_from.data(), d_edges_from, sizeof(int) * E, hipMemcpyDeviceToHost));
        CUCHECK(hipMemcpy(sorted_to.data(), d_edges_to, sizeof(int) * E, hipMemcpyDeviceToHost));

        // Print the sorted edges
        std::cout << "Sorted Edges:" << std::endl;
        for (int i = 0; i < E; ++i) {
            int idx = sorted_index[i];
            std::cout << i << ": (" << sorted_from[idx] << ", " << sorted_to[idx] << ")" << std::endl;
        }
    #endif

    update_first_last_nxt<<<numBlocks, blockSize>>>(
        d_edges_from, 
        d_edges_to, 
        d_first, 
        d_last, 
        d_next, 
        d_index, 
        E);

    CUDA_CHECK(hipDeviceSynchronize(), "Failed to synchronize update_first_last_nxt");

    int *succ = g_bcg_ds.succ;
    int *devRank = g_bcg_ds.devRank;

    cal_succ<<<numBlocks, blockSize>>>(succ, d_next, d_first, d_edges_from, E);
    CUDA_CHECK(hipDeviceSynchronize(), "Failed to synchronize cal_succ");

    #ifdef DEBUG
        std::cout << "successor array before break_cycle_kernel:\n";
        print_device_array(succ, E);
    #endif

    // break cycle_kernel
    numBlocks = (roots_count + blockSize - 1) / blockSize;
    break_cycle_kernel<<<numBlocks, blockSize>>>(d_last, succ, d_roots, roots_count, E);
    CUDA_CHECK(hipDeviceSynchronize(), "Failed to synchronize break_cycle_kernel");

    CudaSimpleListRank(
        devRank, E, succ, 
        g_bcg_ds.notAllDone, g_bcg_ds.devRankNext, g_bcg_ds.devNotAllDone); // these 3 are auxilary ds

    #ifdef DEBUG
        std::cout << "d_first array:\n";
        print_device_array(d_first, N);

        std::cout << "d_last array:\n";
        print_device_array(d_last, N);

        std::cout << "d_next array:\n";
        print_device_array(d_next, E);

        std::cout << "successor array:\n";
        print_device_array(succ, E);

        std::cout << "euler Path array:\n";
        print_device_array(devRank, E);
    #endif

    numBlocks = (E + blockSize - 1) / blockSize;
    find_parent<<<numBlocks, blockSize>>>(E, devRank, d_edges_to, d_edges_from, d_parent);
    CUCHECK(hipDeviceSynchronize());

    // compute level

    // print_device_edges(d_edges_from, E);
    // print_device_edges(d_edges_to, E);

    compute_level(devRank, d_edges_from, d_edges_to, d_parent, g_bcg_ds.devW1Sum, g_bcg_ds.d_level, N, E);
    
    #ifdef DEBUG
        std::cout << "Parent array:" << std::endl;
        print_device_array(d_parent, N);

        std::cout << "devW1Sum array:" << std::endl;
        print_device_array(g_bcg_ds.devW1Sum, E);

        std::cout << "d_level array:" << std::endl;
        print_device_array(g_bcg_ds.d_level, N);
    #endif


    //memset d_first_occ and d_last_occ
    //first_occ of root is 0
    //last_occ of root is E - 1

    numBlocks = (E + blockSize - 1) / blockSize;
    compute_first_last_occ<<<numBlocks, blockSize>>>(
        d_edges_from, 
        d_edges_to, 
        d_first_occ, 
        d_last_occ, 
        g_bcg_ds.d_level,
        devRank, 
        E);
    CUCHECK(hipDeviceSynchronize());

    auto end = std::chrono::high_resolution_clock::now();
    auto dur = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "Eulerian Tour Construction time: " << dur << " ms." << std::endl;

    if(g_verbose) {
        std::cout << "Parent array:" << std::endl;
        print_device_array(d_parent, N);

        std::cout << "d_level array:" << std::endl;
        print_device_array(g_bcg_ds.d_level, N);
        
        std::cout << "First Occurrence array:" << std::endl;
        print_device_array(d_first_occ, N);

        std::cout << "Last Occurrence array:" << std::endl;
        print_device_array(d_last_occ, N);
    }
    
}